#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define MAX_SIZE 1000000 // Taille maximale pour le texte et la clé
#define MAX_KEY_SIZE 1000000


//kernel qui encode le texte
__global__ void encrypt_kernel(const char *cleartext, const char *key, char *ciphertext, int text_len, int key_len) {
    //calcul du numéro de la case sur laquelle l'itération travaille
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < text_len) {
        char c = cleartext[idx];
        //si la lettre à encoder est minuscule
        if (c >= 'a' && c <= 'z') {
            //on ajoute la valeur de la lettre de la clé à la valeur de la lettre à coder
            //on fait ensuite modulo 26 pour avoir la nouvelle lettre codée
            c = ((c - 'a') + (key[idx % key_len] - 'a')) % 26 + 'a';
        } else if (c >= 'A' && c <= 'Z') {
            c = ((c - 'A') + (key[idx % key_len] - 'a')) % 26 + 'A';
        }
        ciphertext[idx] = c;
    }
}

void cipher(const char *cleartext, const char *key, char *ciphertext) {
    //récupération de la taille du texte et de la clé pour les malloc et memcpy
    int cleartext_len = strlen(cleartext);
    int key_len = strlen(key);

    //définition des tableaux de charactère dans le GPU
    char *d_cleartext, *d_key, *d_ciphertext;

    //allocation de la mémoire pour les tableau stockés dans le GPU
    hipMalloc((void **)&d_cleartext, (cleartext_len + 1) * sizeof(char));
    hipMalloc((void **)&d_key, (key_len + 1) * sizeof(char));
    hipMalloc((void **)&d_ciphertext, (cleartext_len + 1) * sizeof(char));

    //copie du texte dans la variable GPU
    hipError_t status =hipMemcpy(d_cleartext, cleartext, (cleartext_len + 1) * sizeof(char), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        printf("Erreur lors de la copie de d_cleartext vers le GPU : %s\n", hipGetErrorString(status));
    }

    //copie de la clé dans la variable GPU
    status =hipMemcpy(d_key, key, (key_len + 1) * sizeof(char), hipMemcpyHostToDevice);
    if (status != hipSuccess) {
        printf("Erreur lors de la copie de d_key vers le GPU : %s\n", hipGetErrorString(status));
    }

    //appel du kernel. la première valeure correspond au nombre de blocs à appeler, la deuxième au nombre de thread par bloc
    encrypt_kernel<<<(cleartext_len + 255) / 256, 256>>>(d_cleartext, d_key, d_ciphertext, cleartext_len, key_len);
    //avec cette fonction, le code CPU va attendre que le code gpu ait fini de tourner
    hipDeviceSynchronize();
    //copie du texte encodé dans la mémoire CPU
    hipMemcpy(ciphertext, d_ciphertext, (cleartext_len + 1) * sizeof(char), hipMemcpyDeviceToHost);


    //libération de la mémoire
    hipFree(d_cleartext);
    hipFree(d_key);
    hipFree(d_ciphertext);

    //ajout du caractère nul à la fin du texte
    ciphertext[cleartext_len] = '\0'; 
}

int main()
{

    char cleartext[MAX_SIZE], key[MAX_KEY_SIZE], ciphertext[MAX_SIZE];

    FILE *plaintextFile = fopen("plaintext.txt", "r");
    if (plaintextFile == NULL)
    {
        perror("Error opening plaintext.txt");
        return 1;
    }
    int cleartextSize = fread(cleartext, 1, MAX_SIZE, plaintextFile);
    fclose(plaintextFile);

    FILE *keyFile = fopen("key.txt", "r");
    if (keyFile == NULL)
    {
        perror("Error opening key.txt");
        return 1;
    }

    int keySize = fread(key, 1, MAX_SIZE, keyFile);
    fclose(keyFile);

    cleartext[cleartextSize] = '\0';
    key[keySize] = '\0';

    //utilisation de cuda event pour calculer le temp d'exécution
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    //récupération de "l'heure" au début du code
    hipEventRecord(start, 0);

    cipher(cleartext, key, ciphertext);
    

    FILE *ciphertextFile = fopen("ciphertext.txt", "w");
    if (ciphertextFile == NULL)
    {
        perror("Error opening ciphertext.txt for writing");
        return 1;
    }
    //récupération de "l'heure" à la fin du code
    hipEventRecord(stop, 0);
    //synchronisation du code avec l'event "stop" pour s'assurer que le temps calculé soit correct
    hipEventSynchronize(stop);
    //calcul du temps d'exécution avec une fonction de la bibliothèque CUDA
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time: %f ms\n", elapsedTime);
    fwrite(ciphertext, 1, strlen(ciphertext), ciphertextFile);

    fclose(ciphertextFile);

    return 0;
}